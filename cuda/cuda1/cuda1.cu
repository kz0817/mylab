
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main (void)
{
    int a = 5;
    int b = 3;
    int c;
    int *x = 0;

    hipMalloc((void**)&x, sizeof(int));
    add<<<1,1>>>(a, b, x);
    hipMemcpy(&c, x, sizeof(int), hipMemcpyDeviceToHost);

    printf("a: %d, b: %d => c: %d\n", a, b, c);
    return 0;
}
