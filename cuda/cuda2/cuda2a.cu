
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <cassert>
using namespace std;

__global__ void add(float *arr1, float *arr2, float *ret, size_t total)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= total)
        return;
    ret[tid] = arr1[tid] + arr2[tid];
}

static void initData(float *arr, const size_t num, const float k)
{
    for (size_t i = 0; i < num; i++)
        arr[i] = k * i + 1;
}

struct Args {
    int numBlocks;
    int numThreads;

    struct Ctx {
        int argc;
        char **argv;
        int idx;

        char *get(void)
        {
            return argv[idx];
        }
    };

    int getNextInt(Ctx &ctx)
    {
        assert(ctx.idx != ctx.argc -1);
        ctx.idx++;
        int param = atoi(ctx.get());
        return param;
    }

    Args(int argc, char *argv[])
    : numBlocks(1),
      numThreads(1)
    {
        Ctx ctx {argc, argv, 0};
        for (ctx.idx = 1; ctx.idx < argc; ctx.idx++) {
            const string arg = ctx.get();
            if (arg == "-b") {
                numBlocks = getNextInt(ctx);
            } else if (arg == "-t") {
                numThreads = getNextInt(ctx);
            }
        }
    }
};

int main(int argc, char *argv[])
{
    const Args args(argc, argv);
    printf("numBlocks :  %d\n", args.numBlocks);
    printf("numThreads: %d\n",  args.numThreads);

    float *arr1, *arr2, *ret;
    float *devArr1, *devArr2, *devRet;

    const size_t numPoints = 8;
    const size_t arrBytes = numPoints * sizeof(float);
    arr1 = (float *)malloc(arrBytes);
    arr2 = (float *)malloc(arrBytes);
    ret  = (float *)malloc(arrBytes);

    initData(arr1, numPoints, 2.1);
    initData(arr2, numPoints, 3.2);

    hipError_t err = hipSuccess;
    err = hipMalloc((void**)&devArr1, arrBytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate devArr1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&devArr2, arrBytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate devArr2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void**)&devRet,  arrBytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate devRet (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devArr1, arr1, arrBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy arr1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devArr2, arr2, arrBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy arr2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    add<<<args.numBlocks, args.numThreads>>>(devArr1, devArr2, devRet, numPoints);
    err = hipGetLastError();
    if (err != hipSuccess) {   
        fprintf(stderr, "Failed to launch add kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(ret, devRet, arrBytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {   
        fprintf(stderr, "Failed to copy devRet (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (size_t i = 0; i < numPoints; i++)
        printf("[%02zd] a: %.3f + b: %.3f => x: %.3f\n", i, arr1[i], arr2[i], ret[i]);

    err = hipFree(devArr1);
    if (err != hipSuccess) {   
        fprintf(stderr, "Failed to free devArr1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(devArr2);
    if (err != hipSuccess) {   
        fprintf(stderr, "Failed to free devArr2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(devRet);
    if (err != hipSuccess) {   
        fprintf(stderr, "Failed to free devRet (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(ret);
    free(arr2);
    free(arr1);
    return 0;
}
