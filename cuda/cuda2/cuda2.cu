
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(float *arr1, float *arr2, float *ret)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    ret[tid] = arr1[tid] + arr2[tid];
}

static void initData(float *arr, const size_t num, const float k)
{
    for (size_t i = 0; i < num; i++)
        arr[i] = k * i + 1;
}

int main (void)
{
    float *arr1, *arr2, *ret;
    float *devArr1, *devArr2, *devRet;

    const size_t numPoints = 8;
    const size_t arrBytes = numPoints * sizeof(float);
    arr1 = (float *)malloc(arrBytes);
    arr2 = (float *)malloc(arrBytes);
    ret  = (float *)malloc(arrBytes);

    initData(arr1, numPoints, 2.1);
    initData(arr2, numPoints, 3.2);

    hipMalloc((void**)&devArr1, arrBytes);
    hipMalloc((void**)&devArr2, arrBytes);
    hipMalloc((void**)&devRet,  arrBytes);

    hipMemcpy(devArr1, arr1, arrBytes, hipMemcpyHostToDevice);
    hipMemcpy(devArr2, arr2, arrBytes, hipMemcpyHostToDevice);

    const size_t numThreads = 4;
    const size_t numBlocks = 2;
    add<<<numBlocks, numThreads>>>(devArr1, devArr2, devRet);
    hipMemcpy(ret, devRet, arrBytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < numPoints; i++)
        printf("[%02zd] a: %.3f + b: %.3f => x: %.3f\n", i, arr1[i], arr2[i], ret[i]);

    hipFree(devArr1);
    hipFree(devArr2);
    hipFree(devRet);

    free(ret);
    free(arr2);
    free(arr1);
    return 0;
}
